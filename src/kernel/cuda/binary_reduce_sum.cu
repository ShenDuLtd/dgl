#include <dgl/runtime/device_api.h>

#include "../../runtime/cuda/cuda_common.h"
#include "./binary_reduce_impl.cuh"
#include "./backward_binary_reduce_impl.cuh"

using minigun::Csr;
using minigun::advance::RuntimeConfig;

namespace dgl {
namespace kernel {
namespace cuda {

// specialization for cusparse

void CusparseCsrmm2(const RuntimeConfig& rtcfg, const Csr& csr, GData<float>* gdata) {
  const int m = csr.row_offsets.length - 1;
  const int k = csr.row_offsets.length - 1;
  const int n = gdata->x_length;
  const int nnz = csr.column_indices.length;
  float alpha = 1.0;
  float beta = 0.0;
  // device
  auto device = runtime::DeviceAPI::Get(rtcfg.ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // allocate cusparse handle if needed
  if (!thr_entry->cusparse_handle) {
    CUSPARSE_CALL(hipsparseCreate(&(thr_entry->cusparse_handle)));
  }
  hipsparseMatDescr_t descr;
  CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
  CUSPARSE_CALL(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CUSPARSE_CALL(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
  //hipsparseScsrmm(
      //thr_entry->cusparse_handle,
      //HIPSPARSE_OPERATION_NON_TRANSPOSE,
      //HIPSPARSE_OPERATION_TRANSPOSE,
      //m, n, k, nnz, alpha,
  LOG(FATAL) << "NOT IMPLEMENTED";
}

void CusparseCsrmm2(const RuntimeConfig& rtcfg, const Csr& csr, GData<double>* gdata) {
  LOG(FATAL) << "NOT IMPLEMENTED";
}

template <>
void CallBinaryReduce<float, DirectId<kDLGPU, int64_t>,
                      SelectSrc, SelectDst,
                      BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(
    const RuntimeConfig& rtcfg,
    const Csr& csr,
    GData<float>* gdata) {
  CusparseCsrmm2(rtcfg, csr, gdata);
}

template <>
void CallBinaryReduce<float, DirectId<kDLGPU, int64_t>,
                      SelectSrc, SelectEdge,
                      BinaryUseLhs<float>, ReduceSum<kDLGPU, float>>(
    const RuntimeConfig& rtcfg,
    const Csr& csr,
    GData<float>* gdata) {
  CusparseCsrmm2(rtcfg, csr, gdata);
}

template <>
void CallBinaryReduce<double, DirectId<kDLGPU, int64_t>,
                      SelectSrc, SelectDst,
                      BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(
    const RuntimeConfig& rtcfg,
    const Csr& csr,
    GData<double>* gdata) {
  CusparseCsrmm2(rtcfg, csr, gdata);
}

template <>
void CallBinaryReduce<double, DirectId<kDLGPU, int64_t>,
                      SelectSrc, SelectEdge,
                      BinaryUseLhs<double>, ReduceSum<kDLGPU, double>>(
    const RuntimeConfig& rtcfg,
    const Csr& csr,
    GData<double>* gdata) {
  CusparseCsrmm2(rtcfg, csr, gdata);
}

// generate definitions

#define REDUCER ReduceSum
#define XPU kDLGPU
#define GETID DirectId

EVAL(GEN_DTYPE, GEN_TARGET, GEN_BINARY_OP, GEN_DEFINE)
EVAL(GEN_BACKWARD_MODE, GEN_DTYPE, GEN_TARGET, GEN_BINARY_OP, GEN_BACKWARD_DEFINE)

}  // namespace cuda
}  // namespace kernel
}  // namespace dgl
