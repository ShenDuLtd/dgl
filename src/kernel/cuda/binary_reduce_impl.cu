#include <dlpack/dlpack.h>
#include <minigun/minigun.h>
#include <dgl/runtime/device_api.h>

#include "../../runtime/cuda/cuda_common.h"
#include "../binary_reduce.h"
#include "./binary_reduce_impl.h"
#include "./utils.cuh"

using dgl::runtime::NDArray;
using minigun::Csr;
using minigun::IntArray1D;

namespace dgl {
namespace kernel {
namespace cuda {
namespace {
inline int64_t ComputeXLength(NDArray feat_array) {
  int64_t ret = 1;
  for (int i = 1; i < feat_array->ndim; ++i) {
    ret *= feat_array->shape[i];
  }
  return ret;
}

inline int64_t NElements(NDArray array) {
  if (IsNoneArray(array)) {
    return 0;
  } else {
    int64_t ret = 1;
    for (int i = 0; i < array->ndim; ++i) {
      ret *= array->shape[i];
    }
    return ret;
  }
}
inline int64_t Prod(const std::vector<int64_t>& vec) {
  int64_t ret = 1;
  for (int64_t v : vec) {
    ret *= v;
  }
  return ret;
}
}  // namespace

template <typename DType, typename Reducer>
GData<DType> AllocGData(hipStream_t stream, int64_t x_len,
                        NDArray lhs_mapping, NDArray rhs_mapping,
                        NDArray lhs_data, NDArray rhs_data,
                        NDArray out_mapping, NDArray out_data) {
  // GData
  GData<DType> gdata;
  gdata.x_length = x_len;
  gdata.lhs_data = static_cast<DType*>(lhs_data->data);
  gdata.rhs_data = static_cast<DType*>(rhs_data->data);
  gdata.out_data = static_cast<DType*>(out_data->data);
  if (!IsNoneArray(lhs_mapping)) {
    gdata.lhs_mapping = static_cast<int64_t*>(lhs_mapping->data);
  }
  if (!IsNoneArray(rhs_mapping)) {
    gdata.rhs_mapping = static_cast<int64_t*>(rhs_mapping->data);
  }
  if (!IsNoneArray(out_mapping)) {
    gdata.out_mapping = static_cast<int64_t*>(out_mapping->data);
  }
  // fill out data with zero values
  utils::Fill(stream, gdata.out_data, NElements(out_data), Zero<Reducer>::value);
  return gdata;
}

void BinaryReduceImpl(
    const std::string& reducer,
    const std::string& op,
    NDArray indptr,
    NDArray indices,
    binary_op::Target lhs,
    binary_op::Target rhs,
    NDArray lhs_mapping,
    NDArray rhs_mapping,
    NDArray lhs_data,
    NDArray rhs_data,
    NDArray out_mapping,
    NDArray out_data) {
  // device
  auto device = runtime::DeviceAPI::Get(out_data->ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // Graph
  Csr csr;
  csr.row_offsets.data = static_cast<mg_int*>(indptr->data);
  csr.row_offsets.length = indptr->shape[0];
  csr.column_indices.data = static_cast<mg_int*>(indices->data);
  csr.column_indices.length = indices->shape[0];
  const int64_t x_len = ComputeXLength(out_data);

  // advance config
  minigun::advance::RuntimeConfig rtcfg;
  rtcfg.ctx = out_data->ctx;
  rtcfg.stream = thr_entry->stream;
  const int nt = utils::FindNumThreads(x_len, 64);
  rtcfg.data_num_threads = nt;
  // XXX(minjie): hard-code to let each thread compute two elements to increase
  //              instruction level parallelism
  rtcfg.data_num_blocks = (x_len + (nt * 2) - 1) / (nt * 2);

  const DLDataType& dtype = out_data->dtype;
  const bool has_indirect =
    !(IsNoneArray(lhs_mapping) && IsNoneArray(rhs_mapping) && IsNoneArray(out_mapping));
  DGL_DTYPE_SWITCH(dtype, DType, {
    REDUCER_SWITCH(reducer, kDLGPU, DType, Reducer, {
      GData<DType> gdata = AllocGData<DType, Reducer>(
          rtcfg.stream, x_len, lhs_mapping, rhs_mapping,
          lhs_data, rhs_data, out_mapping, out_data);
      BINARY_OP_SWITCH(op, DType, BinaryOp, {
        TARGET_SWITCH(lhs, rhs, LeftTarget, RightTarget, {
          if (has_indirect) {
            typedef IndirectId<kDLGPU, int64_t> IdGetter;
            CallBinaryReduce<DType, IdGetter, LeftTarget,
              RightTarget, BinaryOp, Reducer>(rtcfg, csr, &gdata);
          } else {
            typedef DirectId<kDLGPU, int64_t> IdGetter;
            CallBinaryReduce<DType, IdGetter, LeftTarget,
              RightTarget, BinaryOp, Reducer>(rtcfg, csr, &gdata);
          }
        });
      });
    });
    if (reducer == binary_op::kReduceMean) {
      // TODO(minjie): divide
      LOG(FATAL) << "reduce mean is not supported.";
    }
  });
}

template <int NDim, typename DType, typename Reducer>
BcastGData<NDim, DType> AllocBcastGData(
    hipStream_t stream,
    const BcastInfo& info,
    NDArray lhs_mapping, NDArray rhs_mapping,
    NDArray lhs_data, NDArray rhs_data,
    NDArray out_mapping, NDArray out_data) {
  // GData
  BcastGData<NDim, DType> gdata;
  // dim, shape and stride
  gdata.ndim = info.lhs_shape.size();
  std::copy(info.lhs_shape.begin(), info.lhs_shape.end(), gdata.lhs_shape);
  std::copy(info.lhs_stride.begin(), info.lhs_stride.end(), gdata.lhs_stride);
  std::copy(info.rhs_shape.begin(), info.rhs_shape.end(), gdata.rhs_shape);
  std::copy(info.rhs_stride.begin(), info.rhs_stride.end(), gdata.rhs_stride);
  std::copy(info.out_shape.begin(), info.out_shape.end(), gdata.out_shape);
  std::copy(info.out_stride.begin(), info.out_stride.end(), gdata.out_stride);
  gdata.lhs_len = Prod(info.lhs_shape);
  gdata.rhs_len = Prod(info.rhs_shape);
  gdata.out_len = Prod(info.out_shape);
  // data
  gdata.lhs_data = static_cast<DType*>(lhs_data->data);
  gdata.rhs_data = static_cast<DType*>(rhs_data->data);
  gdata.out_data = static_cast<DType*>(out_data->data);
  if (!IsNoneArray(lhs_mapping)) {
    gdata.lhs_mapping = static_cast<int64_t*>(lhs_mapping->data);
  }
  if (!IsNoneArray(rhs_mapping)) {
    gdata.rhs_mapping = static_cast<int64_t*>(rhs_mapping->data);
  }
  if (!IsNoneArray(out_mapping)) {
    gdata.out_mapping = static_cast<int64_t*>(out_mapping->data);
  }
  // fill out data with zero values
  utils::Fill(stream, gdata.out_data, NElements(out_data), Zero<Reducer>::value);
  return gdata;
}

void BinaryReduceBcastImpl(
    const BcastInfo& info,
    const std::string& reducer,
    const std::string& op,
    NDArray indptr,
    NDArray indices,
    binary_op::Target lhs,
    binary_op::Target rhs,
    NDArray lhs_mapping,
    NDArray rhs_mapping,
    NDArray lhs_data,
    NDArray rhs_data,
    NDArray out_mapping,
    NDArray out_data) {
  // device
  auto device = runtime::DeviceAPI::Get(out_data->ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // Graph
  Csr csr;
  csr.row_offsets.data = static_cast<mg_int*>(indptr->data);
  csr.row_offsets.length = indptr->shape[0];
  csr.column_indices.data = static_cast<mg_int*>(indices->data);
  csr.column_indices.length = indices->shape[0];
  const int64_t x_len = ComputeXLength(out_data);

  // advance config
  minigun::advance::RuntimeConfig rtcfg;
  rtcfg.ctx = out_data->ctx;
  rtcfg.stream = thr_entry->stream;
  const int nt = utils::FindNumThreads(x_len, 64);
  rtcfg.data_num_threads = nt;
  // XXX(minjie): hard-code to let each thread compute two elements to increase
  //              instruction level parallelism
  rtcfg.data_num_blocks = (x_len + (nt * 2) - 1) / (nt * 2);

  const DLDataType& dtype = out_data->dtype;
  const bool has_indirect =
    !(IsNoneArray(lhs_mapping) && IsNoneArray(rhs_mapping) && IsNoneArray(out_mapping));
  const int bcast_ndim = info.out_shape.size();
  DGL_DTYPE_SWITCH(dtype, DType, {
    REDUCER_SWITCH(reducer, kDLGPU, DType, Reducer, {
      BCAST_NDIM_SWITCH(bcast_ndim, NDim, {
        BcastGData<NDim, DType> gdata = AllocBcastGData<NDim, DType, Reducer>(
            rtcfg.stream, info, lhs_mapping, rhs_mapping,
            lhs_data, rhs_data, out_mapping, out_data);
        BINARY_OP_SWITCH(op, DType, BinaryOp, {
          TARGET_SWITCH(lhs, rhs, LeftTarget, RightTarget, {
            if (has_indirect) {
              typedef IndirectId<kDLGPU, int64_t> IdGetter;
              CallBinaryReduceBcast<NDim, DType, IdGetter, LeftTarget,
                RightTarget, BinaryOp, Reducer>(rtcfg, csr, &gdata);
            } else {
              typedef DirectId<kDLGPU, int64_t> IdGetter;
              CallBinaryReduceBcast<NDim, DType, IdGetter, LeftTarget,
                RightTarget, BinaryOp, Reducer>(rtcfg, csr, &gdata);
            }
          });
        });
      });
    });
    if (reducer == binary_op::kReduceMean) {
      // TODO(minjie): divide
      LOG(FATAL) << "reduce mean is not supported.";
    }
  });
}

// backward implementations

template <typename DType>
BackwardGData<DType> AllocBackwardGData(
    hipStream_t stream, int64_t x_len,
    NDArray lhs_mapping, NDArray rhs_mapping, NDArray out_mapping,
    NDArray lhs_data, NDArray rhs_data, NDArray out_data, NDArray grad_out_data,
    NDArray grad_lhs_data, NDArray grad_rhs_data) {
  // GData
  BackwardGData<DType> gdata;
  gdata.x_length = x_len;
  gdata.lhs_data = static_cast<DType*>(lhs_data->data);
  gdata.rhs_data = static_cast<DType*>(rhs_data->data);
  gdata.out_data = static_cast<DType*>(out_data->data);
  gdata.grad_out_data = static_cast<DType*>(grad_out_data->data);
  if (!IsNoneArray(grad_lhs_data)) {
    gdata.grad_lhs_data = static_cast<DType*>(grad_lhs_data->data);
    // fill out data with zero values
    utils::Fill(stream, gdata.grad_lhs_data, NElements(grad_lhs_data),
                static_cast<DType>(0));
  }
  if (!IsNoneArray(grad_rhs_data)) {
    gdata.grad_rhs_data = static_cast<DType*>(grad_rhs_data->data);
    // fill out data with zero values
    utils::Fill(stream, gdata.grad_rhs_data, NElements(grad_rhs_data),
                static_cast<DType>(0));
  }
  if (!IsNoneArray(lhs_mapping)) {
    gdata.lhs_mapping = static_cast<int64_t*>(lhs_mapping->data);
  }
  if (!IsNoneArray(rhs_mapping)) {
    gdata.rhs_mapping = static_cast<int64_t*>(rhs_mapping->data);
  }
  if (!IsNoneArray(out_mapping)) {
    gdata.out_mapping = static_cast<int64_t*>(out_mapping->data);
  }
  return gdata;
}

void BackwardBinaryReduceImpl(
    const std::string& reducer,
    const std::string& op,
    NDArray rev_indptr, NDArray rev_indices,
    binary_op::Target lhs, binary_op::Target rhs,
    NDArray lhs_mapping, NDArray rhs_mapping, NDArray out_mapping,
    NDArray lhs_data, NDArray rhs_data, NDArray out_data,
    NDArray grad_out_data,
    NDArray grad_lhs_data, NDArray grad_rhs_data) {
  // device
  auto device = runtime::DeviceAPI::Get(out_data->ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // Graph
  Csr csr;
  csr.row_offsets.data = static_cast<mg_int*>(rev_indptr->data);
  csr.row_offsets.length = rev_indptr->shape[0];
  csr.column_indices.data = static_cast<mg_int*>(rev_indices->data);
  csr.column_indices.length = rev_indices->shape[0];
  const int64_t x_len = ComputeXLength(out_data);

  // advance config
  minigun::advance::RuntimeConfig rtcfg;
  rtcfg.ctx = out_data->ctx;
  rtcfg.stream = thr_entry->stream;
  const int nt = utils::FindNumThreads(x_len, 64);
  rtcfg.data_num_threads = nt;
  // XXX(minjie): hard-code to let each thread compute two elements to increase
  //              instruction level parallelism
  rtcfg.data_num_blocks = (x_len + (nt * 2) - 1) / (nt * 2);

  const DLDataType& dtype = out_data->dtype;
  const bool has_indirect =
    !(IsNoneArray(lhs_mapping) && IsNoneArray(rhs_mapping) && IsNoneArray(out_mapping));
  const bool req_lhs = !IsNoneArray(grad_lhs_data);
  const bool req_rhs = !IsNoneArray(grad_rhs_data);
  BACKWARD_MODE_SWITCH(req_lhs, req_rhs, Mode, {
    DGL_DTYPE_SWITCH(dtype, DType, {
      BackwardGData<DType> gdata = AllocBackwardGData<DType>(
          rtcfg.stream, x_len,
          lhs_mapping, rhs_mapping, out_mapping,
          lhs_data, rhs_data, out_data, grad_out_data,
          grad_lhs_data, grad_rhs_data);
      //REDUCER_SWITCH(reducer, kDLGPU, DType, Reducer, {
        typedef ReduceSum<kDLGPU, DType> Reducer;
        BINARY_OP_SWITCH(op, DType, BinaryOp, {
          TARGET_SWITCH(lhs, rhs, LeftTarget, RightTarget, {
            if (has_indirect) {
              typedef IndirectId<kDLGPU, int64_t> IdGetter;
              CallBackwardBinaryReduce<Mode, DType, IdGetter, LeftTarget,
                RightTarget, BinaryOp, Reducer>(rtcfg, csr, &gdata);
            } else {
              typedef DirectId<kDLGPU, int64_t> IdGetter;
              CallBackwardBinaryReduce<Mode, DType, IdGetter, LeftTarget,
                RightTarget, BinaryOp, Reducer>(rtcfg, csr, &gdata);
            }
          });
        });
      //});
      if (reducer == binary_op::kReduceMean) {
        // TODO(minjie): divide
        LOG(FATAL) << "reduce mean is not supported.";
      }
    });
  });
}

template <int NDim, typename DType>
BackwardBcastGData<NDim, DType> AllocBackwardBcastGData(
    hipStream_t stream,
    const BcastInfo& info,
    NDArray lhs_mapping, NDArray rhs_mapping, NDArray out_mapping,
    NDArray lhs, NDArray rhs, NDArray out, NDArray grad_out,
    NDArray grad_lhs, NDArray grad_rhs) {
  // GData
  BackwardBcastGData<NDim, DType> gdata;
  // dim, shape and stride
  gdata.ndim = info.lhs_shape.size();
  gdata.lhs_len = Prod(info.lhs_shape);
  gdata.rhs_len = Prod(info.rhs_shape);
  gdata.out_len = Prod(info.out_shape);
  std::copy(info.lhs_shape.begin(), info.lhs_shape.end(), gdata.lhs_shape);
  std::copy(info.lhs_stride.begin(), info.lhs_stride.end(), gdata.lhs_stride);
  std::copy(info.rhs_shape.begin(), info.rhs_shape.end(), gdata.rhs_shape);
  std::copy(info.rhs_stride.begin(), info.rhs_stride.end(), gdata.rhs_stride);
  std::copy(info.out_shape.begin(), info.out_shape.end(), gdata.out_shape);
  std::copy(info.out_stride.begin(), info.out_stride.end(), gdata.out_stride);
  // mappings
  if (!IsNoneArray(lhs_mapping)) {
    gdata.lhs_mapping = static_cast<int64_t*>(lhs_mapping->data);
  }
  if (!IsNoneArray(rhs_mapping)) {
    gdata.rhs_mapping = static_cast<int64_t*>(rhs_mapping->data);
  }
  if (!IsNoneArray(out_mapping)) {
    gdata.out_mapping = static_cast<int64_t*>(out_mapping->data);
  }
  // data
  gdata.lhs_data = static_cast<DType*>(lhs->data);
  gdata.rhs_data = static_cast<DType*>(rhs->data);
  gdata.out_data = static_cast<DType*>(out->data);
  gdata.grad_out_data = static_cast<DType*>(grad_out->data);
  if (!IsNoneArray(grad_lhs)) {
    gdata.grad_lhs_data = static_cast<DType*>(grad_lhs->data);
    // fill out data with zero values
    utils::Fill(stream, gdata.grad_lhs_data, NElements(grad_lhs),
                static_cast<DType>(0));
  }
  if (!IsNoneArray(grad_rhs)) {
    gdata.grad_rhs_data = static_cast<DType*>(grad_rhs->data);
    // fill out data with zero values
    utils::Fill(stream, gdata.grad_rhs_data, NElements(grad_rhs),
                static_cast<DType>(0));
  }
  return gdata;
}

void BackwardBinaryReduceBcastImpl(
    const BcastInfo& info,
    const std::string& reducer,
    const std::string& op,
    NDArray rev_indptr, NDArray rev_indices,
    binary_op::Target lhs_tgt, binary_op::Target rhs_tgt,
    NDArray lhs_mapping, NDArray rhs_mapping, NDArray out_mapping,
    NDArray lhs, NDArray rhs, NDArray out, NDArray grad_out,
    NDArray grad_lhs, NDArray grad_rhs) {
  // device
  auto device = runtime::DeviceAPI::Get(out->ctx);
  auto* thr_entry = runtime::CUDAThreadEntry::ThreadLocal();
  // Graph
  Csr csr;
  csr.row_offsets.data = static_cast<mg_int*>(rev_indptr->data);
  csr.row_offsets.length = rev_indptr->shape[0];
  csr.column_indices.data = static_cast<mg_int*>(rev_indices->data);
  csr.column_indices.length = rev_indices->shape[0];
  const int64_t x_len = ComputeXLength(out);

  // advance config
  minigun::advance::RuntimeConfig rtcfg;
  rtcfg.ctx = out->ctx;
  rtcfg.stream = thr_entry->stream;
  const int nt = utils::FindNumThreads(x_len, 64);
  rtcfg.data_num_threads = nt;
  // XXX(minjie): hard-code to let each thread compute two elements to increase
  //              instruction level parallelism
  rtcfg.data_num_blocks = (x_len + (nt * 2) - 1) / (nt * 2);

  const DLDataType& dtype = out->dtype;
  const bool has_indirect =
    !(IsNoneArray(lhs_mapping) && IsNoneArray(rhs_mapping) && IsNoneArray(out_mapping));
  const int bcast_ndim = info.out_shape.size();
  const bool req_lhs = !IsNoneArray(grad_lhs);
  const bool req_rhs = !IsNoneArray(grad_rhs);
  BACKWARD_MODE_SWITCH(req_lhs, req_rhs, Mode, {
    DGL_DTYPE_SWITCH(dtype, DType, {
      BCAST_NDIM_SWITCH(bcast_ndim, NDim, {
        BackwardBcastGData<NDim, DType> gdata = AllocBackwardBcastGData<NDim, DType>(
            rtcfg.stream, info,
            lhs_mapping, rhs_mapping, out_mapping,
            lhs, rhs, out, grad_out,
            grad_lhs, grad_rhs);
        //REDUCER_SWITCH(reducer, kDLGPU, DType, Reducer, {
        typedef ReduceSum<kDLGPU, DType> Reducer;
          BINARY_OP_SWITCH(op, DType, BinaryOp, {
            TARGET_SWITCH(lhs_tgt, rhs_tgt, LeftTarget, RightTarget, {
              if (has_indirect) {
                typedef IndirectId<kDLGPU, int64_t> IdGetter;
                CallBackwardBinaryReduceBcast<Mode, NDim, DType, IdGetter, LeftTarget,
                  RightTarget, BinaryOp, Reducer>(rtcfg, csr, &gdata);
              } else {
                typedef DirectId<kDLGPU, int64_t> IdGetter;
                CallBackwardBinaryReduceBcast<Mode, NDim, DType, IdGetter, LeftTarget,
                  RightTarget, BinaryOp, Reducer>(rtcfg, csr, &gdata);
              }
            });
          });
        //});
      });
      if (reducer == binary_op::kReduceMean) {
        // TODO(minjie): divide
        LOG(FATAL) << "reduce mean is not supported.";
      }
    });
  });
}

}  // namespace cuda
}  // namespace kernel
}  // namespace dgl
